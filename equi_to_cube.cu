#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <cmath>
#include <hip/hip_runtime.h>
#include <string>

#define PI 3.14159265358979323846

using namespace std;

// Helper math functions
__device__ void faceDirection(int faceIdx, float u, float v, float3& dir) {
    float x = 2.0f * u - 1.0f;
    float y = 2.0f * v - 1.0f;
    switch (faceIdx) {
        case 0: dir = make_float3(1, -y, -x); break;    // +X (right)
        case 1: dir = make_float3(-1, -y, x); break;    // -X (left)
        case 2: dir = make_float3(x, 1, y); break;      // +Y (top)
        case 3: dir = make_float3(x, -1, -y); break;    // -Y (bottom)
        case 4: dir = make_float3(x, -y, 1); break;     // +Z (front)
        case 5: dir = make_float3(-x, -y, -1); break;   // -Z (back)
    }
    float len = sqrtf(dir.x * dir.x + dir.y * dir.y + dir.z * dir.z);
    dir.x /= len; dir.y /= len; dir.z /= len;
}

__device__ void dirToEquirect(float3 dir, int width, int height, int& u, int& v) {
    float theta = atan2f(dir.x, -dir.z);
    float phi = asinf(dir.y);
    float uf = (theta + PI) / (2 * PI);
    float vf = (phi + PI / 2) / PI;
    u = int(uf * width);
    v = int(vf * height);
    u = min(max(u, 0), width - 1);
    v = min(max(v, 0), height - 1);
}

__device__ void rotateCoords(int& x, int& y, int faceSize, int rotation) {
    int tx = x, ty = y;
    switch (rotation) {
        case 90:
            x = ty;
            y = faceSize - 1 - tx;
            break;
        case 180:
            x = faceSize - 1 - tx;
            y = faceSize - 1 - ty;
            break;
        case 270:
            x = faceSize - 1 - ty;
            y = tx;
            break;
        default:
            break;
    }
}

__global__ void EquirectToCubeKernel(uchar3* input, uchar3* output, int eqWidth, int eqHeight, int faceSize) {
    int face = blockIdx.z;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= faceSize || y >= faceSize || face >= 6) return;

    float u = (x + 0.5f) / faceSize;
    float v = (y + 0.5f) / faceSize;
    float3 dir;
    faceDirection(face, u, v, dir);

    int uE, vE;
    dirToEquirect(dir, eqWidth, eqHeight, uE, vE);
    uchar3 color = input[vE * eqWidth + uE];

    int outX = 0, outY = 0, rot = 0;
    switch (face) {
        case 0: outX = 2 * faceSize; outY = faceSize; rot = 270; break; // +X
        case 1: outX = 0;            outY = faceSize; rot = 90;  break; // -X
        case 2: outX = faceSize;     outY = 0;         rot = 180; break; // +Y (top)
        case 3: outX = faceSize;     outY = 2 * faceSize; rot = 0; break; // -Y (bottom)
        case 4: outX = faceSize;     outY = faceSize;   rot = 0;  break; // +Z
        case 5: outX = 3 * faceSize; outY = faceSize;   rot = 180; break; // -Z
    }

    int tx = x, ty = y;
    rotateCoords(tx, ty, faceSize, rot);
    int outIdx = (outY + ty) * faceSize * 4 + (outX + tx);
    output[outIdx] = color;
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cout << "Usage: ./equirect_to_cube <equirectangular_image>" << std::endl;
        return -1;
    }

    cv::Mat img = cv::imread(argv[1], cv::IMREAD_COLOR);
    if (img.empty()) {
        std::cerr << "Error loading image!" << std::endl;
        return -1;
    }

    int eqWidth = img.cols;
    int eqHeight = img.rows;
    int faceSize = eqWidth / 4;

    size_t inputSize = eqWidth * eqHeight * sizeof(uchar3);
    size_t outputSize = (faceSize * 4) * (faceSize * 3) * sizeof(uchar3);

    uchar3* d_input;
    uchar3* d_output;
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_output, outputSize);
    hipMemcpy(d_input, img.ptr<uchar3>(), inputSize, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((faceSize + 15) / 16, (faceSize + 15) / 16, 6);
    EquirectToCubeKernel<<<grid, block>>>(d_input, d_output, eqWidth, eqHeight, faceSize);
    hipDeviceSynchronize();

    cv::Mat result(faceSize * 3, faceSize * 4, CV_8UC3);
    hipMemcpy(result.ptr<uchar3>(), d_output, outputSize, hipMemcpyDeviceToHost);

    cv::imwrite("cube_map_output.jpg", result);

    hipFree(d_input);
    hipFree(d_output);

    std::cout << "Saved: cube_map_output.jpg" << std::endl;
    return 0;
}
